#include "hip/hip_runtime.h"
#include "../Common/cuMatrix.h"
#include "FullyConnect.h"
#include <cstdlib>

__global__ void relu(float *inout, float *bias, int rows, int cols) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (j >= cols || i >= rows) return;
    inout[i * cols + j] = fmaxf(0.0, inout[i * cols + j] + bias[i]);
}

__global__ void relu_grad(float *pre_grad, float *output, int rows, int cols) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (j >= cols || i >= rows) return;
    if (output[i * cols + j] <= 0)
        pre_grad[i * cols + j] = 0;
}

__global__ void bias_grad(float *pre_grad, float *output, int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= rows) return;
    output[i] = 0;
    for (int k = 0; k < cols; k++) {
        output[i] += pre_grad[i * cols + k];
    }
}

void FullyConnect::initRandom() {
    this->w = new cuMatrix<float>(this->units, this->inputs->rows);
    this->b = new cuMatrix<float>(this->units, 1);

    // this->w->setAllRandom(-1, 1);
    // this->b->setAllRandom(-1, 1);

    for (int j = 0; j < this->inputs->rows; j++) {
        for (int i = 0; i < this->units; i++) {
            this->w->set(i, j, j + 1);
        }
    }

    for (int i = 0; i < this->units; i++) {
        this->b->set(i, 0, i);
    }
}

void FullyConnect::feedforward() {
    this->w->toGpu();
    this->b->toGpu();
    this->inputs->toGpu();
    this->outputs->toGpu();
    matrixMul(this->w, this->inputs, this->outputs);

    dim3 blockDim(16, 16, 1);
    dim3 gridDim((this->outputs->cols + blockDim.x - 1) / blockDim.x,
                 (this->outputs->rows + blockDim.y - 1) / blockDim.y);
    relu << < blockDim, gridDim >> > (outputs->getDev(), this->b->getDev(), this->units, this->batch);
}

cuMatrix<float> *FullyConnect::getOutputs() {
    return this->outputs;
}

void FullyConnect::printParameter() {
    printf("weights:\n");
    this->w->printHost();
    printf("bias:\n");
    this->b->printHost();
    printf("inputs:\n");
    this->inputs->printHost();
    printf("outputs:\n");
    this->outputs->printHost();
    printf("bias gradient\n");
    b_grad->printHost();
    printf("inputs gradient\n");
    inputs_grad->printHost();
    printf("weights gradient\n");
    w_grad->printHost();
}

void FullyConnect::backpropagation(cuMatrix<float> *pre_grad) {
    dim3 blockDim_r(16, 16, 1);
    dim3 gridDim_r((outputs->cols + blockDim_r.x - 1) / blockDim_r.x,
                   (outputs->rows + blockDim_r.y - 1) / blockDim_r.y);
    relu_grad << < blockDim_r, gridDim_r >> > (pre_grad->getDev(), outputs->getDev(), outputs->rows, outputs->cols);
    printf("after relu\n");
    pre_grad->printHost();
    dim3 blockDim_b(256);
    dim3 gridDim_b((b->rows + blockDim_b.x - 1) / blockDim_b.x);
    bias_grad << < blockDim_b, gridDim_b >> > (pre_grad->getDev(), b_grad->getDev(), pre_grad->rows, pre_grad->cols);
    matrixTranspose(inputs_grad);
    matrixMulTA(pre_grad, w, inputs_grad);
    matrixTranspose(inputs_grad);
    matrixMulTB(pre_grad, inputs, w_grad);
    updateWeight();
}

void FullyConnect::getGrad() {

}

void FullyConnect::updateWeight() {
    matrixSub(w, w_grad, w, lambda);
    matrixSub(b, b_grad, b, lambda);
}

void FullyConnect::clearGrad() {

}





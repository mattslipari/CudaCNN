#include "hip/hip_runtime.h"
#include "../Common/cuMatrix.h"
#include "FullyConnect.h"
#include <cstdlib>

__global__ void relu(float *inout, float *bias, int rows, int cols) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (j >= cols || i >= rows) return;
    inout[i * cols + j] = fmaxf(0.0, inout[i * cols + j] + bias[i]);
}

__global__ void relu_grad(float *pre_grad, float *output, int rows, int cols) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (j >= cols || i >= rows) return;
    if (output[i * cols + j] <= 0)
        pre_grad[i * cols + j] = 0;
}

__global__ void bias_grad(float *pre_grad, float *output, int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= rows) return;
    output[i] = 0;
    for (int k = 0; k < cols; k++) {
        output[i] = pre_grad[i * cols + k];
    }
}

void FullyConnect::initRandom() {
    this->w = new cuMatrix<float>(this->units, this->inputs->rows);
    this->b = new cuMatrix<float>(this->units, 1);

    // this->w->setAllRandom(-1, 1);
    // this->b->setAllRandom(-1, 1);

    for (int j = 0; j < this->inputs->rows; j++) {
        for (int i = 0; i < this->units; i++) {
            this->w->set(i, j, j + 1);
        }
    }

    for (int i = 0; i < this->units; i++) {
        this->b->set(i, 0, i);
    }
}

void FullyConnect::feedforward() {
    this->w->toGpu();
    this->b->toGpu();
    this->inputs->toGpu();
    this->outputs->toGpu();
    matrixMul(this->w, this->inputs, this->outputs);

    dim3 blockDim(16, 16, 1);
    dim3 gridDim((this->outputs->cols + blockDim.x - 1) / blockDim.x,
                 (this->outputs->rows + blockDim.y - 1) / blockDim.y);
    relu << < blockDim, gridDim >> > (outputs->getDev(), this->b->getDev(), this->units, this->batch);
}

cuMatrix<float> *FullyConnect::getOutputs() {
    return this->outputs;
}

void FullyConnect::printParameter() {
    this->w->toCpu();
    this->b->toCpu();
    printf("weights:\n");
    this->w->printHost();
    printf("bias:\n");
    this->b->printHost();
    this->inputs->toCpu();
    printf("inputs:\n");
    this->inputs->printHost();
    this->outputs->toCpu();
    printf("outputs:\n");
    this->outputs->printHost();
}

void FullyConnect::backpropagation(cuMatrix<float> *pre_grad) {
    dim3 blockDim_r(16, 16, 1);
    dim3 gridDim_r((outputs->cols + blockDim.x - 1) / blockDim.x,
                 (outputs->rows + blockDim.y - 1) / blockDim.y);
    relu_grad << < blockDim_r, gridDim_r >> > (pre_grad->getDev(), outputs->getDev(), outputs->rows, outputs->cols);
    
    dim3 blockDim_b(256);
    dim3 gridDim_b((b->rows + blockDim.x - 1) / blockDim.x);
    bias_grad << < blockDim_b, gridDim_b >> > (pre_grad->getDev(), b_grad->getDev(), pre_grad->rows, pre_grad->cols);
    
    matrixMulTA(pre_grad, w, inputs_grad);
    matrixTranspose(inputs_grad);
    matrixMulTB(pre_grad, inputs, w_grad);
}

void FullyConnect::getGrad() {

}

void FullyConnect::updateWeight() {

}

void FullyConnect::clearGrad() {

}





#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "hipblas.h"

#define IDX2C(i, j, ld) (((j)*(ld))+(i))

__global__ void relu(float *input, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    input[idx] = fmaxf(0.0, input[idx]);
}

__global__ void softmax(float *A, float *p, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float sum = 0;
        for (int i = 0; i < n; i++) {
            p[i] = expf(A[i]);
            sum = sum + p[i];
        }
        p[idx] = logf(p[idx] / sum);
    }
}

static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}

void fully_connected(hipblasHandle_t *handle, float *x, float *y, float *w, int n, int m) {
    float alpha = 1;
    float beta = 0;
    hipblasStatus_t stat;
    stat=hipblasSgemv(*handle, HIPBLAS_OP_N,
                n, m,
                &alpha,
                w, n,
                x, 1,
                &beta,
                y, 1);
    dim3 blockDim(16);
    printf("%s\n", _cudaGetErrorEnum(stat));

    float *tmp = (float *) malloc(n * sizeof(*y));
    hipblasGetVector(n, sizeof(*y), y, 1, tmp, 1); //cp d_c->c printf("c after Sgemm :\n");
    for (int i = 0; i < n; i++) {
        printf("%7.0f", tmp[i]); //print c after Sgemm
    }

    relu <<< blockDim, 1 >>> (y, n);

}

void forward(float *x, float *w1, float *w2, float *y, float *loss, int n, int m) {
    // hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);
    dim3 blockDim(16);
    float *output1;
    float *output2;
    float *predict;

    hipMalloc((void **) &output1, n * sizeof(*x));
    hipMalloc((void **) &output2, n * sizeof(*x));
    hipMalloc((void **) &predict, n * sizeof(*x));

    fully_connected(&handle, x, output1, w1, n, m);
    fully_connected(&handle, output1, output2, w2, n, n);

    //float index;
    //hipblasIsamin(handle,n,output,1,&index);
    softmax << < blockDim, 1 >> > (output2, predict, n);

    // stat = hipblasSdot(handle,n,predict,1,y,1,loss);
    hipblasSdot(handle, n, predict, 1, y, 1, loss);
}

int main() {
    float *x;
    float *y;
    float *w1;
    float *w2;

    int n = 5;
    int m = 3;
    float loss;

    x = (float *) malloc(m * sizeof(*x));
    y = (float *) malloc(n * sizeof(*y));
    w1 = (float *) malloc(n * m * sizeof(*w1));
    w2 = (float *) malloc(n * n * sizeof(*w2));

    for (int j = 0; j < m; j++) {
        x[j] = 1;
    }
    for (int j = 0; j < n * m; j++) {
        w1[j] = 1;
    }

    for (int j = 0; j < n * n; j++) {
        w2[j] = 1;
    }

    float *c_x = NULL;
    float *c_y = NULL;
    float *c_w1 = NULL;
    float *c_w2 = NULL;

    hipMalloc((void **) &c_x, m * sizeof(*x));
    hipMalloc((void **) &c_y, n * sizeof(*y));
    hipMalloc((void **) &c_w1, n * m * sizeof(*w1));
    hipMalloc((void **) &c_w2, n * n * sizeof(*w2));

    hipblasSetMatrix(n, m, sizeof(float), (void *) w1, n, (void *) c_w1, n);
    hipblasSetMatrix(n, n, sizeof(float), (void *) w2, n, (void *) c_w2, n);

    hipblasSetVector(m, sizeof(float), (void *) x, 1, (void *) c_x, 1);
    hipblasSetVector(n, sizeof(float), (void *) y, 1, (void *) c_y, 1);

    forward(c_x, c_w1, c_w2, c_y, &loss, n, m);
    return 0;
}

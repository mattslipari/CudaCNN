#include "hip/hip_runtime.h"
/*
Modified from
https://github.com/zhxfl/CUDA-CNN
*/

#include "cuMatrix.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "CycleTimer.h"


hipblasHandle_t &getHandle() {
    static hipblasHandle_t handle = NULL;
    if (handle == NULL) {
        hipblasStatus_t stat;
        stat = hipblasCreate(&handle);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
            printf("init: CUBLAS initialization failed\n");
            exit(0);
        }
    }
    return handle;
}

__global__ void elementwiseMul(float *x, float *y, float *z, int rows, int cols) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (j >= cols || i >= rows) return;
    z[i * cols + j] = x[i * cols + j] * y[i * cols + j];
}

void matrixElementWiseMul(cuMatrix<float> *x, cuMatrix<float> *y, cuMatrix<float> *z) {
    if (x->cols != y->cols || z->cols != x->cols || x->rows != y->rows || x->rows != z->rows) {
        printf("matrix elementwise multiply invalid dim\n");
        exit(0);
    }
    dim3 blockDim(16, 16, 1);
    dim3 gridDim((x->cols + blockDim.x - 1) / blockDim.x,
                 (x->rows + blockDim.y - 1) / blockDim.y);
    elementwiseMul << < blockDim, gridDim >> > (x->getDev(), y->getDev(), z->getDev(), x->rows, x->cols);

}

/*Matrix Concatenation*/
/*z = [x;y]*/
void matrixConcat(cuMatrix<float> *x, cuMatrix<float> *y, cuMatrix<float> *z) {
    if (x->cols != y->cols || z->cols != x->cols || z->rows != x->rows + y->rows) {
        printf("matrix concat invalid dim\n");
        exit(0);
    }

    float *res = z->getDev();
    hipMemcpy(res, x->getDev(), x->rows * x->cols * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(&res[x->rows * x->cols], y->getDev(), y->rows * y->cols * sizeof(float), hipMemcpyDeviceToDevice);
}

/*Matrix Split*/
/*y = x[1:row][:] z = x[row:end][:]*/
void matrixSplit(cuMatrix<float> *x, cuMatrix<float> *y, cuMatrix<float> *z) {
    if (x->cols != y->cols || x->cols != z->cols || x->rows != y->rows + z->rows) {
        printf("matrix split invalid dim\n");
        exit(0);
    }

    hipMemcpy(y->getDev(), x->getDev(), y->rows * y->cols * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(z->getDev(), &(x->getDev())[y->rows * y->cols], z->rows * z->cols * sizeof(float),
               hipMemcpyDeviceToDevice);
}

/*matrix transpose*/
/*x = T(x)*/
double matrixTranspose(cuMatrix<float> *x) {
    float alpha = 1.0;
    float beta = 0.0;
    float *y;
    hipblasHandle_t handle = getHandle();
    double overallStartTime = CycleTimer::currentSeconds();
    hipMalloc(&y, x->rows * x->cols * sizeof(float));
    hipMemcpy(y, x->getDev(), x->rows * x->cols * sizeof(float), hipMemcpyDeviceToDevice);


    hipblasSgeam(handle,
                HIPBLAS_OP_T,
                HIPBLAS_OP_N,
                x->rows, x->cols,
                &alpha,
                y, x->cols,
                &beta,
                NULL, x->rows,
                x->getDev(), x->rows);

    int temp_r = x->rows;
    x->rows = x->cols;
    x->cols = temp_r;
    hipFree(y);
    double overallEndTime = CycleTimer::currentSeconds();
    return overallEndTime - overallStartTime;
}

__global__ void matrixTransKernel(float *A, int rows, int cols) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (j >= cols || i >= rows) return;
    float tmp = A[i * cols + j];
    A[i * cols + j] = A[j * cols + i];
    A[j * cols + i] = tmp;
}

double matrixTranspose2(cuMatrix<float> *x) {
    double overallStartTime = CycleTimer::currentSeconds();
    dim3 blockDim(16, 16);
    dim3 gridDim((x->cols + blockDim.x - 1) / blockDim.x,
                 (x->rows + blockDim.y - 1) / blockDim.y);

    matrixTransKernel << < blockDim, gridDim >> > (x->getDev(), x->rows,x->cols);
    hipDeviceSynchronize();
    int temp_r = x->rows;
    x->rows = x->cols;
    x->cols = temp_r;
    double overallEndTime = CycleTimer::currentSeconds();
    return overallEndTime - overallStartTime;
}

__global__ void matrixSubKernel(float *A, float *B, float *C, float lambda, int N) {
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    int COL = blockIdx.x * blockDim.x + threadIdx.x;

    if (ROW < N && COL < N) {
        C[ROW * N + COL] = A[ROW * N + COL] + lambda * B[ROW * N + COL];
    }
}

double matrixSub2(cuMatrix<float> *x, cuMatrix<float> *y, cuMatrix<float> *z, float lambda) {
    double overallStartTime = CycleTimer::currentSeconds();
    lambda = -lambda;
    dim3 blockDim(16, 16);
    int N = x->rows;
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x,
                 (N + blockDim.y - 1) / blockDim.y);

    matrixSubKernel << < blockDim, gridDim >> > (x->getDev(), y->getDev(), z->getDev(), lambda, N);
    hipDeviceSynchronize();
    double overallEndTime = CycleTimer::currentSeconds();
    return overallEndTime - overallStartTime;
}

double matrixSub(cuMatrix<float> *x, cuMatrix<float> *y, cuMatrix<float> *z, float lambda) {
    lambda = -lambda;
    float alpha = 1.0;
    hipblasStatus_t stat;
    hipblasHandle_t handle = getHandle();
    double overallStartTime = CycleTimer::currentSeconds();
    stat = hipblasSgeam(handle,
                       HIPBLAS_OP_N,
                       HIPBLAS_OP_N,
                       x->cols, y->rows,
                       &alpha,
                       x->getDev(), x->cols,
                       &lambda,
                       y->getDev(), y->cols,
                       z->getDev(), z->cols);
    double overallEndTime = CycleTimer::currentSeconds();
    return overallEndTime - overallStartTime;
    //hipStreamSynchronize(0);
//    getLastCudaError("matrixSub");
//    if (stat != HIPBLAS_STATUS_SUCCESS) {
//        printf("matrixSub hipblasSgemm error\n");
//        hipFree(x->getDev());
//        hipFree(y->getDev());
//        hipFree(z->getDev());
//        exit(0);
//    }
}

/*matrix multiply*/
/*z = x * y*/
double matrixMul(cuMatrix<float> *x, cuMatrix<float> *y, cuMatrix<float> *z) {
    if (x->cols != y->rows || z->rows != x->rows || z->cols != y->cols) {
        printf("matrix mul chanels != 1\n");
        exit(0);
    }
    float alpha = 1.0;
    float beta = 0.0;
    hipblasStatus_t stat;
    hipblasHandle_t handle = getHandle();
    double overallStartTime = CycleTimer::currentSeconds();
    stat = hipblasSgemm(
            handle,
            HIPBLAS_OP_N,
            HIPBLAS_OP_N,
            y->cols,
            x->rows,
            y->rows,
            &alpha,
            y->getDev(),
            y->cols,
            x->getDev(),
            x->cols,
            &beta,
            z->getDev(),
            z->cols);
    double overallEndTime = CycleTimer::currentSeconds();
    return overallEndTime - overallStartTime;
    //hipStreamSynchronize(0);
    //getLastCudaError("matrixMul");
//    if (stat != HIPBLAS_STATUS_SUCCESS) {
//        printf("matrixMul hipblasSgemm error\n");
//        hipFree(x->getDev());
//        hipFree(y->getDev());
//        hipFree(z->getDev());
//        exit(0);
//    }
}

__global__ void matrixMultiplicationKernel(float *A, float *B, float *C, int N) {

    int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    int COL = blockIdx.x * blockDim.x + threadIdx.x;

    float tmpSum = 0;

    if (ROW < N && COL < N) {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < N; i++) {
            tmpSum += A[ROW * N + i] * B[i * N + COL];
        }
    }
    C[ROW * N + COL] = tmpSum;
}

double matrixMul2(cuMatrix<float> *x, cuMatrix<float> *y, cuMatrix<float> *z) {
    double overallStartTime = CycleTimer::currentSeconds();
    int N = x->rows;
    //printf("%d\n", N);
    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x,
                 (N + blockDim.y - 1) / blockDim.y);

    matrixMultiplicationKernel << < blockDim, gridDim >> > (x->getDev(), y->getDev(), z->getDev(), N);
    hipDeviceSynchronize();
    double overallEndTime = CycleTimer::currentSeconds();
    return overallEndTime - overallStartTime;
}


/*z = T(x) * y*/
void matrixMulTA(cuMatrix<float> *x, cuMatrix<float> *y, cuMatrix<float> *z) {
    if (x->rows != y->rows || z->rows != x->cols || z->cols != y->cols) {
        printf("matrix mul chanels != 1\n");
        exit(0);
    }
    hipblasStatus_t stat;
    float alpha = 1.0;
    float beta = 0.0;
    stat = hipblasSgemm(
            getHandle(),
            HIPBLAS_OP_N,
            HIPBLAS_OP_T,
            y->cols,
            x->cols,
            y->rows,
            &alpha,
            y->getDev(),
            y->cols,
            x->getDev(),
            x->cols,
            &beta,
            z->getDev(),
            z->cols);
    hipStreamSynchronize(0);
    getLastCudaError("matrixMulTA");
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("matrixMulTA hipblasSgemm error\n");
        exit(0);
    }
}

/*z = x * T(y)*/
void matrixMulTB(cuMatrix<float> *x, cuMatrix<float> *y, cuMatrix<float> *z) {
    if (x->cols != y->cols || z->rows != x->rows || z->cols != y->rows) {
        printf("matrix mul chanels != 1\n");
        exit(0);
    }
    hipblasStatus_t stat;
    float alpha = 1.0;
    float beta = 0.0;
    stat = hipblasSgemm(
            getHandle(),
            HIPBLAS_OP_T,
            HIPBLAS_OP_N,
            y->rows,
            x->rows,
            y->cols,
            &alpha,
            y->getDev(),
            y->cols,
            x->getDev(),
            x->cols,
            &beta,
            z->getDev(),
            z->cols);
    hipStreamSynchronize(0);
    getLastCudaError("matrixMulTB");
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("matrixMulTB hipblasSgemm error\n");
        exit(0);
    }
}